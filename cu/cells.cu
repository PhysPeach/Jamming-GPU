#include "../cuh/cells.cuh"

namespace PhysPeach{
    //cells
    void createCells(Cells *cells, double L){
        cells->numOfCellsPerSide = (int)(L/(2. * a_max));
        if(cells->numOfCellsPerSide < 3){
            cells->numOfCellsPerSide = 3;
        }
        double buf = 3.;
        cells->Nc = (int)(buf * (double)Np/ (double)powInt(cells->numOfCellsPerSide, D));

        int NoC = powInt(cells->numOfCellsPerSide, D)*cells->Nc;
        hipMalloc((void**)&cells->cell_dev, NoC * sizeof(int));
        return;
    }

    void deleteCells(Cells *cells){
        hipFree(cells->cell_dev);
        return;
    }

    void increaseNc(Cells *cells){
        cells->Nc = (int)(1.4 * cells->Nc);
        int NoC = powInt(cells->numOfCellsPerSide, D)*cells->Nc;
        hipFree(cells->cell_dev);
        hipMalloc((void**)&cells->cell_dev, NoC * sizeof(int));
        return;
    }
    //lists
}