#include "hip/hip_runtime.h"
#include "../testcuh/cells_test.cuh"

namespace PhysPeach{
    //cellsTest
    void createCellsTest(){

        Cells cells;
        createCells(&cells, 3.);
        assert(cells.numOfCellsPerSide == 3);
        assert(cells.Nc == (int)(3. * (double)Np/ (double)powInt(3, D)));
        deleteCells(&cells);

        createCells(&cells, 40.);
        assert(cells.numOfCellsPerSide == 12);
        assert(cells.Nc == (int)(3. * (double)Np/ (double)powInt(12, D)));
        deleteCells(&cells);

        return;
    }

    void increaseNcTest(){
        Cells cells;
        createCells(&cells, 40.);

        assert(cells.Nc == (int)(3. * (double)Np/ (double)powInt(12, D)));
        increaseNc(&cells);
        assert(cells.Nc == (int)(1.4 * (int)(3. * (double)Np/ (double)powInt(12, D))));
        deleteCells(&cells);

        return;
    }

    void updateCellsTest(){

        Cells cells;

        if(Np <= 100){
            //test it in small Np
            double *x;
            x = (double*)malloc(D*Np*sizeof(double));
            for(int par1 = 0; par1 < D*Np; par1++){
                x[par1] = 1.;
            }
            createCells(&cells, 10.);
            updateCells(&cells, 10., x);
            assert(cells.Nc == Np+1);
            deleteCells(&cells);
            free(x);
        }

        Particles p;
        createParticles(&p);
        double L = pow(p.packing/Phi_init, 1./(double)D);
        createCells(&cells, L);
        updateCells(&cells, L, p.x_dev);
        deleteCells(&cells);
        deleteParticles(&p);

        return;
    }
    //listsTest
}