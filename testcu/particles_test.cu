#include "hip/hip_runtime.h"
#include "../testcuh/particles_test.cuh"
#include <fstream>

namespace PhysPeach{
    void createParticlesTest(){

        Particles p;
        createParticles(&p);
        hipMemcpy(p.x, p.x_dev, D * Np * sizeof(double), hipMemcpyDeviceToHost);

        double diamav = 0.;
        double xav[D];
        for(int d = 0; d < D; d++){
            xav[d] = 0.;
        }
        for(int par1 = 0; par1 < Np; par1++){
            diamav += p.diam[par1];
            for(int d = 0; d < D; d++){
                xav[d] += p.x[d*Np+par1];
            }
        }

        diamav /= Np;
        assert(0.99 < diamav && diamav < 1.01);

        double L = pow(p.packing/Phi_init, 1./(double)D);
        for(int d = 0; d < D; d++){
            xav[d] /= Np * L;
            assert(-0.01 < xav[d] && xav[d] < 0.01);
        }
        deleteParticles(&p);
        return;
    }

    void powerParticlesTest(){
        Particles p;
        double power;

        createParticles(&p);

        for(int par1 = 0; par1 < D*Np; par1++){
            p.v[par1] = 3.;
        }
        hipMemcpy(p.v_dev, p.v, D * Np * sizeof(double), hipMemcpyHostToDevice);
        for(int par1 = 0; par1 < D*Np; par1++){
            p.v[par1] = 2.;
        }
        hipMemcpy(p.f_dev, p.v, D * Np * sizeof(double), hipMemcpyHostToDevice);

        power = powerParticles(&p);
        assert(5.99 * D*Np < power && power < 6.01 * D*Np);

        for(int par1 = 0; par1 < D*Np; par1++){
            p.v[par1] = 1.;
        }
        hipMemcpy(p.v_dev, p.v, D * Np * sizeof(double), hipMemcpyHostToDevice);
        for(int par1 = 0; par1 < D*Np; par1++){
            p.v[par1] = (double)par1;
        }
        hipMemcpy(p.f_dev, p.v, D * Np * sizeof(double), hipMemcpyHostToDevice);

        power = powerParticles(&p);
        assert((double)(D*Np * (D*Np - 1)/2) - 0.1 < power && power < (double)(D*Np * (D*Np - 1)/2) + 0.1);

        deleteParticles(&p);
        
        return;
    }

    void convergedFireTest(){
        Particles p;
        bool converged;

        createParticles(&p);

        converged = convergedFire(&p);
        assert(converged);

        double f = 1.0e-10 * D*Np;

        f = 1.0e-10 * D*Np;
        hipMemcpy(&p.f_dev[0], &f, sizeof(double), hipMemcpyHostToDevice);
        converged = convergedFire(&p);
        assert(!converged);

        f = 1.0e-12 * D*Np;
        hipMemcpy(&p.f_dev[0], &f, sizeof(double), hipMemcpyHostToDevice);
        converged = convergedFire(&p);
        assert(converged);

        f = 3.0e-12 * D*Np;
        hipMemcpy(&p.f_dev[D*Np - 1], &f, sizeof(double), hipMemcpyHostToDevice);
        converged = convergedFire(&p);
        assert(!converged);

        deleteParticles(&p);

        return;
    }

    void modifyVelocitiesTest(){
        int NB = (Np + NT - 1)/NT;
        Particles p;

        createParticles(&p);

        double s = 1.;
        hipMemcpy(&p.v_dev[0], &s, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(&p.f_dev[Np], &s, sizeof(double), hipMemcpyHostToDevice);
        modifyVelocities<<<NB, NT>>>(p.v_dev, p.f_dev, 0.2, Np);
        hipMemcpy(&s, &p.v_dev[0], sizeof(double), hipMemcpyDeviceToHost);
        assert(0.79 < s && s < 0.81);
        hipMemcpy(&s, &p.v_dev[Np], sizeof(double), hipMemcpyDeviceToHost);
        assert(0.19 < s && s < 0.21);

        modifyVelocities<<<NB, NT>>>(p.v_dev, p.f_dev, 0.3, Np);
        hipMemcpy(&s, &p.v_dev[0], sizeof(double), hipMemcpyDeviceToHost);
        assert(0.55 < s && s < 0.57);
        hipMemcpy(&s, &p.v_dev[Np], sizeof(double), hipMemcpyDeviceToHost);
        assert(0.37 < s && s < 0.39);

        deleteParticles(&p);
        return;
    }
}